#include "Device.cuh"

#include <hip/hip_runtime.h>

#include <stdexcept>

using namespace std;

namespace iki {
	Device::Device(int device) {
		hipError_t cudaStatus;
		if (hipSuccess != (cudaStatus = hipSetDevice(device)))
			throw runtime_error(hipGetErrorString(cudaStatus));
	}

	Device::~Device() noexcept {
		hipDeviceReset();
	}
} /*iki*/