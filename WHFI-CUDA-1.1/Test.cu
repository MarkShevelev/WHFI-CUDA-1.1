#include "hip/hip_runtime.h"
#include "TwoDimensionalMultithreadDiffusion.cuh"
#include "Device.h"
#include "DeviceMemory.h"
#include "DataTable.h"
#include "UniformGrid.h"
#include "UniformGridIO.h"
#include "DataTableDeviceHelper.h"
#include "HostGrid.h"
#include "HostGridIO.h"

#include <iostream>
#include <fstream>

void v_field_init(iki::grid::test::HostGrid<float> &vdf_grid);
void along_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid);
void perp_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid);

int main() {
	using namespace std;
	using namespace iki;
	using namespace table;
	using namespace grid;
	try {
		unsigned vparall_size = 256, vperp_size = 512;
		test::Space<float> v_space{ test::Axis<float>{ -15.f, 1.e-3f }, test::Axis<float>{ 0.f, 1.e-3f } };
		test::Space<float> v_space_transposed{ test::Axis<float>{ 0.f, 1.e-3f }, test::Axis<float>{ -15.f, 1.e-3f } };
		test::HostGrid<float> vdf_grid(v_space, vparall_size, vperp_size);
		test::HostGrid<float> vperp_dfc_grid(v_space, vparall_size, vperp_size);
		test::HostGrid<float> vparall_dfc_grid(v_space_transposed, vperp_size, vparall_size);

		v_field_init(vdf_grid);
		along_dfc_field_init(vperp_dfc_grid);
		perp_dfc_field_init(vparall_dfc_grid);

		Device device(0);

		unsigned const row_count = vparall_size, row_size = vperp_size, field_size = row_size*row_count;
		DeviceMemory dev_memory(field_size * 9 * sizeof(float));
		float *x_prev = (float *)dev_memory;
		float *x_next = x_prev + field_size;
		float *x_tmp = x_next + field_size;
		float *along_dfc = x_tmp + field_size;
		float *perp_dfc = along_dfc + field_size;
		float *a = perp_dfc + field_size;
		float *b = a + field_size;
		float *c = b + field_size;
		float *d = c + field_size;

		hipMemcpy(x_prev, vdf_grid.table.hData.data(), field_size * sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(x_next, vdf_grid.table.hData.data(), field_size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(along_dfc, vperp_dfc_grid.table.hData.data(), field_size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(perp_dfc, vparall_dfc_grid.table.hData.data(), field_size * sizeof(float), hipMemcpyHostToDevice);

		diffusion::TwoDimensionalMultithreadDiffusion<32u, 256u, float> diffusion_solver(row_count, row_size, a, b, c, d, x_prev, x_next, x_tmp, along_dfc, 1.0f, perp_dfc, 1.0f);
		for (unsigned iter_cnt = 0; iter_cnt != 1000; ++iter_cnt)
			diffusion_solver.step();

		hipMemcpy(vdf_grid.table.hData.data(), x_prev, field_size * sizeof(float), hipMemcpyDeviceToHost);
		{
			ofstream ascii_os;
			ascii_os.exceptions(ios::badbit | ios::failbit);
			ascii_os.precision(7); ascii_os.setf(ios::fixed, ios::floatfield);
			ascii_os.open("./data/one-dimensional-sin-test.txt");
			ascii_os << vdf_grid;
		}

	}
	catch (exception &ex) {
		cout << ex.what() << endl;
	}

	return 0;
}

void v_field_init(iki::grid::test::HostGrid<float> &vdf_grid) {
	auto &table = vdf_grid.table;

	for (unsigned prp_idx = 0; prp_idx != table.row_size; ++prp_idx) {
		float val = std::sin(2.f * 3.1415926535f / 128 * prp_idx);
		for (unsigned prl_idx = 0; prl_idx != table.row_count; ++prl_idx)
			table(prl_idx, prp_idx) = val * std::sin(2.f * 3.1415926535f / 128 * prl_idx);
	}

	for (unsigned prp_idx = 0; prp_idx != table.row_size; ++prp_idx)
		table(0, prp_idx) = table(table.row_count - 1, prp_idx) = 0.f;

	for (unsigned prl_idx = 0; prl_idx != table.row_count; ++prl_idx)
		table(prl_idx,0) = table(prl_idx, table.row_size - 1) = 0.f;
}

void along_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid) {
	auto &table = dfc_grid.table;

	for (unsigned row_idx = 0; row_idx != table.row_count; ++row_idx)
		for (unsigned elm_idx = 0; elm_idx != table.row_size; ++elm_idx)
			table(row_idx,elm_idx) = 1.f;
}

void perp_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid) {
	auto &table = dfc_grid.table;

	for (unsigned row_idx = 0; row_idx != table.row_count; ++row_idx)
		for (unsigned elm_idx = 0; elm_idx != table.row_size; ++elm_idx)
			table(row_idx, elm_idx) = 1.f;
}