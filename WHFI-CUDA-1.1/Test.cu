#include "hip/hip_runtime.h"
#include "TwoDimensionalMultithreadDiffusion.cuh"
#include "Device.h"
#include "DeviceMemory.h"
#include "HostGrid.h"
#include "HostGridIO.h"
#include "HostManagedDeviceTable.cuh"
#include "HostDeviceTransfer.cuh"
#include "HostTableTranspose.h"

#include <iostream>
#include <fstream>

void v_field_init(iki::grid::test::HostGrid<float> &vdf_grid);
void along_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid);
void perp_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid);

int main() {
	using namespace std;
	using namespace iki;
	using namespace table;
	using namespace grid;
	namespace gt = iki::grid::test;
	namespace tt = iki::table::test;
	try {
		unsigned vparall_size = 256, vperp_size = 512;
		gt::Space<float> v_space{ gt::Axis<float>{ -15.f, 1.e-3f }, gt::Axis<float>{ 0.f, 1.e-3f } };
		gt::Space<float> v_space_transposed{ gt::Axis<float>{ 0.f, 1.e-3f }, gt::Axis<float>{ -15.f, 1.e-3f } };
		gt::HostGrid<float> vdf_grid(v_space, vparall_size, vperp_size);
		gt::HostGrid<float> vperp_dfc_grid(v_space, vparall_size, vperp_size);
		gt::HostGrid<float> vparall_dfc_grid(v_space_transposed, vperp_size, vparall_size);

		v_field_init(vdf_grid);
		along_dfc_field_init(vperp_dfc_grid);
		perp_dfc_field_init(vparall_dfc_grid);

		Device device(0);

		unsigned const row_count = vparall_size, row_size = vperp_size, field_size = row_size*row_count;

		diffusion::TwoDimensionalMultithreadDiffusion<32u, 256u, float> 
			diffusion_solver(
				vdf_grid.table,
				vparall_dfc_grid.table, 1.0f,
				vperp_dfc_grid.table, 1.0f
			);

		for (unsigned iter_cnt = 0; iter_cnt != 1000; ++iter_cnt)
			diffusion_solver.step();

		{
			gt::HostGrid<float> output_grid(v_space, vparall_size, vperp_size);
			device_to_host_transfer(diffusion_solver.x_prev, output_grid.table);

			ofstream ascii_os;
			ascii_os.exceptions(ios::badbit | ios::failbit);
			ascii_os.precision(7); ascii_os.setf(ios::fixed, ios::floatfield);
			ascii_os.open("./data/one-dimensional-sin-test.txt");
			ascii_os << output_grid;
		}

	}
	catch (exception &ex) {
		cout << ex.what() << endl;
	}

	return 0;
}

void v_field_init(iki::grid::test::HostGrid<float> &vdf_grid) {
	auto &table = vdf_grid.table;

	for (unsigned prp_idx = 0; prp_idx != table.row_size; ++prp_idx) {
		float val = std::sin(2.f * 3.1415926535f / 128 * prp_idx);
		for (unsigned prl_idx = 0; prl_idx != table.row_count; ++prl_idx)
			table(prl_idx, prp_idx) = val * std::sin(2.f * 3.1415926535f / 128 * prl_idx);
	}

	for (unsigned prp_idx = 0; prp_idx != table.row_size; ++prp_idx)
		table(0, prp_idx) = table(table.row_count - 1, prp_idx) = 0.f;

	for (unsigned prl_idx = 0; prl_idx != table.row_count; ++prl_idx)
		table(prl_idx,0) = table(prl_idx, table.row_size - 1) = 0.f;
}

void along_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid) {
	auto &table = dfc_grid.table;

	for (unsigned row_idx = 0; row_idx != table.row_count; ++row_idx)
		for (unsigned elm_idx = 0; elm_idx != table.row_size; ++elm_idx)
			table(row_idx,elm_idx) = 1.f;
}

void perp_dfc_field_init(iki::grid::test::HostGrid<float> &dfc_grid) {
	auto &table = dfc_grid.table;

	for (unsigned row_idx = 0; row_idx != table.row_count; ++row_idx)
		for (unsigned elm_idx = 0; elm_idx != table.row_size; ++elm_idx)
			table(row_idx, elm_idx) = 1.f;
}