#include "hip/hip_runtime.h"
#include "thomson_sweep.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <stdexcept>
#include <iostream>
#include <fstream>
#include <vector>

template <typename T>
__device__ void initial_matrix(T *a, T *b, T *c, T *d, T *x, unsigned size) {
	for (unsigned offset = 0; offset != size; ++offset) {
		a[offset] = 1.f;
		b[offset] = 3.f;
		c[offset] = 1.f;
		d[offset] = 5.f;
		x[offset] = 0;
	}

	d[0] = d[size - 1] = 4.f;
}

template <typename T>
__global__ void thomson_sweep_single_thread_test_kernel(T *a, T *b, T *c, T *d, T *x, unsigned size) {
	initial_matrix(a, b, c, d, x, size);
	iki::math::device::thomson_sweep(a, b, c, d, x, size);
}

int main() {
	using namespace std;
	try {
		hipError_t cudaStatus;
		if (hipSuccess != (cudaStatus = hipSetDevice(0)))
			throw runtime_error(hipGetErrorString(cudaStatus));

		unsigned size = 1024;
		void *dev_memory = NULL;
		cudaStatus = hipMalloc(&dev_memory, sizeof(float) * size * 5);
		if (hipSuccess != cudaStatus)
			throw runtime_error(hipGetErrorString(cudaStatus));

		float *a = (float*)dev_memory;
		float *b = a + size;
		float *c = b + size;
		float *d = c + size;
		float *x = d + size;

		dim3 threads_dim(1), block_dim(1);
		thomson_sweep_single_thread_test_kernel <<<threads_dim, block_dim>>> (a,b,c,d,x,size);

		vector<float> result(size);
		if (hipSuccess != (cudaStatus = hipMemcpy(result.data(), x, size * sizeof(float), hipMemcpyDeviceToHost)))
			throw runtime_error(hipGetErrorString(cudaStatus));

		{
			ofstream ascii_out("./data/result.txt");
			ascii_out.precision(7); ascii_out.setf(ios::fixed, ios::floatfield);
			for (auto const &x : result)
				ascii_out << x << '\n';
		}
			
	}
	catch (exception const &e) {
		cout << e.what() << endl;
	}


	return 0;
}